// Reduction example for parallel programming reading course
//
// Auther: Frederik Andersen
//
// Setup of code and comment is modified from CUDA example addKernel
// Reduce kernel is taken from http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
//

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

hipError_t reduce(int *h_in, int *h_out, unsigned int size);
void createArray(const int size, int* array);
void createResult(int* array, int* result, const int size);


// Reducion 
__global__ void reduce_kernel(int * d_out, int * d_in)
{
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	sdata[tid] = d_in[i] + d_in[i + blockDim.x];
	__syncthreads();
	// do reduction in shared mem

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

int main()
{
	const int arraySize = 256;
	int h_in[arraySize] = { 0 };
	int h_out[1] = { 0 };
	int serialResult[1] = { 0 };

	createArray(arraySize, h_in);
	createResult(h_in, serialResult, arraySize);

    // Add vectors in parallel.
    hipError_t cudaStatus = reduce(h_in, h_out, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("Reduce of %d elements resulted in = {%d}\n", arraySize, h_out[0]);
	if (h_out[0] == serialResult[0])
		printf("Test Passed\n");
	else
		printf("Test Failed, result should be %d but is %d\n", serialResult[0], h_out[0]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	int lol;
	std::cin >> lol;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t reduce(int *h_in, int *h_out,  unsigned int size)
{
    int *d_in = 0;
    int *d_out = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_in, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_out, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_out, h_out, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	reduce_kernel <<<1, size, size*sizeof(int)>>>(d_out, d_in);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduce_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduce_kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(d_out);
    hipFree(d_in);
    
    return cudaStatus;
}

void createArray(const int size, int* array)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = i;
	}
}

void createResult(int* array, int* result, const int size)
{
	for (int i = 0; i < size; i++)
	{
		result[0] += array[i];
	}
}