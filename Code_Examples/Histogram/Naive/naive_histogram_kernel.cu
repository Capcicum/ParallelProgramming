#include "hip/hip_runtime.h"
// Blelloch scan example for parallel programming reading course
//
// Auther: Frederik Andersen
//
// Setup of code and comment is modified from CUDA example addWithCuda
// Scan kernel is taken from https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
//

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define COMPUTE_BIN(n) n && 0x01

hipError_t histogram(int *h_in, int *h_out, unsigned int bins, unsigned int size);
void createArray(const int size, int* array);
void serialVersion(int* h_in, int* h_out, unsigned int bins, unsigned int size);
bool checkResults(int* array1, int* array2, unsigned int size);


// Blelloch  scan 
__global__ void naive_histo_kernel(int *d_bins, const int *d_in, const int BIN_COUNT)
{
	int myId = threadIdx.x;
	int myItem = d_in[myId];
	int myBin = COMPUTE_BIN(myItem);
	atomicAdd(&(d_bins[myBin]), 1);
}

int main()
{
	const int arraySize = 256;
	const int bins = 2;
	int h_in[arraySize] = { 0 };
	int h_out[bins] = { 0 };
	int serialResult[bins] = { 0 };

	createArray(arraySize, h_in);

	// Scan in serial.
	serialVersion(h_in, serialResult, bins, arraySize);

	// Scan in parallel.
	hipError_t cudaStatus = histogram(h_in, h_out, bins, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Histogram of %d elements with %d bins\n", arraySize, bins);
	if (checkResults(h_out, serialResult, bins))
		printf("Test Passed\n");
	else
		printf("Test Failed\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogram(int *h_in, int *h_out, unsigned int bins, unsigned int size)
{
	int *d_in = 0;
	int *d_out = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_in, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_out, bins * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_out, h_out, bins * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	naive_histo_kernel<< <1, size>> >(d_out, d_in, bins);
	cudaStatus = hipDeviceSynchronize();
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "scan_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scan_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(h_out, d_out, bins * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(d_out);
	hipFree(d_in);

	return cudaStatus;
}

void createArray(const int size, int* array)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = i;
	}
}

void serialVersion(int* h_in, int* h_out, unsigned int bins, unsigned int size)
{
	for (int i = 0; i < bins; i++) 
	{
		h_out[i] = 0;
	}
	for (int i = 0; i < size; i++) 
	{
		h_out[COMPUTE_BIN(h_in[i])]++;
	}
}

bool checkResults(int* array1, int* array2, unsigned int size)
{
	bool result = true;
	for (unsigned int i = 0; i < size; i++)
	{
		if (array1[i] != array2[i])
		{
			result = false;
			return result;
		}
	}
	return result;
}