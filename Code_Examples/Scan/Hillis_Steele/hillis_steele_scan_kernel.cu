#include "hip/hip_runtime.h"
// Hillis Steele scan example for parallel programming reading course
//
// Auther: Frederik Andersen
//
// Setup of code and comment is modified from CUDA example addWithCuda
// Scan kernel is taken from https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
//

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

hipError_t scan(int *h_in, int *h_out, unsigned int size);
void createArray(const int size, int* array);
void serialVersion(int* h_in, int* h_out, unsigned int size);
bool checkResults(int* array1, int* array2, unsigned int size);


// Hillis Steele scan 
__global__ void scan_kernel(int *d_out, int *d_in, int size)
{
	extern __shared__ int temp[]; // allocated on invocation
	int thid = threadIdx.x;

	// load input into shared memory.
	// This is exclusive scan, so shift right by one and set first elt to 0
	temp[thid] = (thid > 0) ? d_in[thid - 1] : 0;
	__syncthreads();
	for (int offset = 1; offset < size; offset *= 2)
	{
		if (thid >= offset)
			temp[thid] += temp[thid - offset];
		else
			temp[thid] = temp[ thid];
		__syncthreads();
	}
	d_out[thid] = temp[thid]; // write output
}

int main()
{
	const int arraySize = 8;
	int h_in[arraySize] = { 0 };
	int h_out[arraySize] = { 0 };
	int serialResult[arraySize] = { 0 };

	createArray(arraySize, h_in);

	// Scan in serial.
	serialVersion(h_in, serialResult, arraySize);

	// Scan in parallel.
	hipError_t cudaStatus = scan(h_in, h_out, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "scan failed!");
		return 1;
	}

	printf("Scan of %d elements", arraySize);
	if (checkResults(h_out, serialResult, arraySize))
		printf("Test Passed\n");
	else
		printf("Test Failed\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	int lol;
	std::cin >> lol;
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t scan(int *h_in, int *h_out, unsigned int size)
{
	int *d_in = 0;
	int *d_out = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_in, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_out, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_out, h_out, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	scan_kernel << <1, size, size * sizeof(int) >> >(d_out, d_in, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "scan_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scan_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(h_out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(d_out);
	hipFree(d_in);

	return cudaStatus;
}

void createArray(const int size, int* array)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = i;
	}
}

void serialVersion(int* h_in, int* h_out, unsigned int size)
{
	h_out[0] = 0; // since this is a prescan, not a scan
	for (unsigned int j = 1; j < size; ++j)
	{
		h_out[j] = h_in[j - 1] + h_out[j - 1];
	}
}

bool checkResults(int* array1, int* array2, unsigned int size)
{
	bool result = true;
	for (unsigned int i = 0; i < size; i++)
	{
		if (array1[i] != array2[i])
		{
			result = false;
			return result;
		}
	}
	return result;
}