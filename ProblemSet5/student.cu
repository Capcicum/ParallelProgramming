#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <>


__global__
void yourHisto(unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
	__shared__ unsigned ar[1024];
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	for(int i = 0 ; i < 200 ; ++i)
		atomicAdd(&(ar[vals[idx + i * 51200]]), 1);
	__syncthreads();
	atomicAdd(&histo[threadIdx.x], ar[threadIdx.x]);
}

void computeHistogram(unsigned int* const d_vals,	//INPUT
                      unsigned int* const d_histo,		//OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
	yourHisto << <50, 1024 >> >(d_vals, d_histo, numElems);
}

